#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/find.h>

#include <cstdio>
#include <iostream>
#include <cstring>
#include <vector>

using namespace std;

__global__ void fnSearch(char *str, char *key, int *res)
{
	*res = -1;
	if(str[threadIdx.x] == *key)
		*res = threadIdx.x;
}

void mtf(vector<char> word)
{
	//Parallel initialisation of character set	
	thrust::device_vector<char> d_list(256);
	thrust::sequence(d_list.begin(), d_list.begin() + 256);
	thrust::host_vector<char> list(256);
	thrust::device_vector<char> d_word(word.size());
	thrust::device_vector<int> dRes;
	int counter;
	thrust::device_vector<char>::iterator iter;
	thrust::host_vector<char> h_word(word.size());
	char ch;
/*	h_word = word;
	d_word = h_word;
	h_word = d_word;
*/	thrust::copy(word.begin(), word.end(), d_word.begin());
	thrust::device_vector<char> temp(256);
	for (counter = 0; counter < word.size(); counter++)
	{
		//Scan for character on cpu
		iter = thrust::find(d_list.begin(), d_list.end(), d_word[counter]);

		//Shifting of the character set in parallel		
		thrust::copy(d_list.begin(), iter - 1, temp.begin());
		thrust::copy(temp.begin(), iter - 1, d_list.begin() + 1);

		d_list[0] = d_word[counter];
		thrust::copy(d_list.begin(), d_list.end(), list.begin());
	}
	for (counter = 0; counter <= word.size(); counter++)
	{
		ch = list[counter];		
		cout << counter << "\t" << ch << endl;
	}
}

int main(int argc, char *argv[])
{
	if (argc != 2)
	{
		cout << "Usage: mtf.out STRING_INPUT" << endl;
		exit(1);
	}

	int len = strlen(argv[1]);
	char *arg = new char(len);
	strcpy(arg, argv[1]);
	vector<char> word(arg, arg + sizeof(arg));
	/*while ( *(arg) != '\0' )
	{
		word.push_back(*(arg++));
	}*/
	mtf(word);
	return 0;
}

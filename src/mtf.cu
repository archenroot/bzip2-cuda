#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/find.h>

#include <cstdio>
#include <iostream>
#include <cstring>
#include <vector>

using namespace std;

__global__ void fnSearch(char *str, char *key, int *res)
{
	*res = -1;
	if(str[threadIdx.x] == *key)
		*res = threadIdx.x;
}

void mtf(vector<char> word)
{
	//Parallel initialisation of character set	
	thrust::device_vector<char> d_list(256);
	thrust::sequence(d_list.begin(), d_list.begin() + 256);
	thrust::host_vector<char> list(256);
	thrust::device_vector<char> d_word(word.size());
	thrust::device_vector<int> dRes;
	int counter;
	thrust::device_vector<char>::iterator iter;
	thrust::host_vector<char> h_word(word.size());
	char ch;
	h_word = word;
	d_word = h_word;

	int i;

	for (counter = 0; counter < word.size(); counter++)
	{
		thrust::device_vector<char> temp(256);
		thrust::copy(list.begin(), list.end(), d_list.begin());
		//Scan for character on cpu
		iter = thrust::find(d_list.begin(), d_list.end(), d_word[counter]);
		h_word[0] = d_word[counter];

		//Shifting of the character set in parallel
	
		thrust::copy(d_list.begin(), d_list[0] + iter - 1, temp.begin());
		thrust::copy(temp.begin(), temp.end(), d_list.begin() + 1);
		d_list[0] = h_word[0];

		thrust::copy(d_list.begin(), d_list.end(), list.begin());
	}

	for (counter = 0; counter < word.size(); counter++)
	{
		ch = list[counter];		
		cout << counter << "\t" << ch << endl;
	}
}

int main(int argc, char *argv[])
{
	if (argc != 2)
	{
		cout << "Usage: mtf.out STRING_INPUT" << endl;
		exit(1);
	}

	int len = strlen(argv[1]);
	vector<char> word(argv[1], argv[1] + len);
	/*while ( *(arg) != '\0' )
	{
		word.push_back(*(arg++));
	}*/
	mtf(word);
	return 0;
}

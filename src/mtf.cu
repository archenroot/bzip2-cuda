#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/find.h>

#include <cstdio>
#include <iostream>
#include <cstring>

using namespace std;

__global__ void fnSearch(char *str, char *key, int *res)
{
	*res = -1;
	if(str[threadIdx.x] == *key)
		*res = threadIdx.x;
}

void mtf(char* word)
{
	//Parallel initialisation of character set	
	thrust::device_vector<char> d_list(256);
	thrust::sequence(d_list.begin(), d_list.begin() + 256);
	thrust::host_vector<char> list(256);
	thrust::device_vector<char> d_word(strlen(word));
	thrust::device_vector<int> dRes;
	int counter, index;
	d_word = word;
//	hipMemcpy(d_word, word, sizeof(char) *strlen(word), hipMemcpyHostToDevice);

	for (counter = 0; counter < word.length(); counter++)
	{
		//Scan for character on cpu
		
		fnSearch<<<1, 256>>>(d_list, d_word[counter], dRes);
/*		thrust::copy(d_list.begin(), d_list.end(), list.begin());
		for (index = 0; ; index++)
		{
			if (word[counter] == list[index])
				break;
		}
*/
		//Shifting of the character set in parallel
		thrust::device_vector<char> temp(256);
		thrust::copy(d_list.begin(), d_list.begin() + index - 1, temp.begin());
		thrust::copy(temp.begin(), temp.begin() + index - 1, d_list.begin() + 1);

//		thrust::copy(d_list.begin(), d_list.end(), list.begin());
		d_list[0] = d_word[counter];
//		d_list = list;
	}
	for (counter = 0; counter <= word.length(); counter++)
	{
		char ch = list[counter];		
		cout << counter << "\t" << ch << endl;
	}
}

int main(int argc, char *argv[])
{
	if (argc != 2)
	{
		cout << "Usage: mtf.out STRING_INPUT" << endl;
		exit(1);
	}

	char* word = new char(strlen(argv[1]);
	word = argv[1];
	mtf(word);
	return 0;
}

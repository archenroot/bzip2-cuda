#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/find.h>

#include <cstdio>
#include <iostream>
#include <cstring>
#include <vector>

using namespace std;

__global__ void fnSearch(char *str, char *key, int *res)
{
	*res = -1;
	if(str[threadIdx.x] == *key)
		*res = threadIdx.x;
}

void mtf(vector<char> word)
{
	//Parallel initialisation of character set	
	thrust::device_vector<char> d_list(256);
	thrust::sequence(d_list.begin(), d_list.begin() + 256);
	thrust::host_vector<char> list(256);
	thrust::device_vector<char> d_word(word.size());
	thrust::device_vector<int> dRes;
	int counter, index;
	thrust::copy(word.begin(), word.end(), d_word.begin());

	for (counter = 0; counter < word.size(); counter++)
	{
		//Scan for character on cpu
		
		fnSearch<<<1, 256>>>(d_list, d_word[counter], dRes);

		//Shifting of the character set in parallel
		thrust::device_vector<char> temp(256);
		thrust::copy(d_list.begin(), d_list.begin() + index - 1, temp.begin());
		thrust::copy(temp.begin(), temp.begin() + index - 1, d_list.begin() + 1);

		d_list[0] = d_word[counter];
		thrust::copy(d_list.begin(), d_list.end(), list.begin());
	}
	for (counter = 0; counter <= word.size(); counter++)
	{
		char ch = list[counter];		
		cout << counter << "\t" << ch << endl;
	}
}

int main(int argc, char *argv[])
{
	if (argc != 2)
	{
		cout << "Usage: mtf.out STRING_INPUT" << endl;
		exit(1);
	}

	int len = strlen(argv[1]);
	char *arg = new char(len);
	strcpy(arg, argv[1]);
	
	vector<char> word(len);
	while ( *(arg) != '\0' )
	{
		word.push_back(*(arg++));
	}
	mtf(word);
	return 0;
}

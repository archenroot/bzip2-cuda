#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/fill.h>

#include <cstdio>
#include <iostream>
#include <cstring>
#include <iterator>

#define POOL_SZ (10*1024*1024)

using namespace std;

////////////////////DEVICE_STRING STARTS
class device_string
{
public:
	int cstr_len;
	char* raw;
	thrust::device_ptr<char> cstr;

	static char* pool_raw;
	static thrust::device_ptr<char> pool_cstr;
	static thrust::device_ptr<char> pool_top;

	// Sets the variables up the first time its used.
	__host__ static void init()
	{
    		static bool v = true;
		if( v )
		{
			v = false;

			pool_cstr = thrust::device_malloc(POOL_SZ);
			pool_raw  = (char*)raw_pointer_cast( pool_cstr );
			pool_top = pool_cstr;
		}
        }
	// Destructor for device variables used.
	__host__ static void fin()
	{
		init();
		thrust::device_free(pool_cstr);
	}

	// Parametrized constructor to copy one device_string to another.
	__host__ device_string( const device_string& s )
	{
		cstr_len = s.cstr_len;
		raw = s.raw;
		cstr = s.cstr;
	}

	// Parametrized constructor to copy a std::string to device_string type
	__host__ device_string( const std::string& s )
	{
		cstr_len = s.length();
		init();
		cstr = pool_top;
		pool_top += cstr_len+1;
		raw = (char *) raw_pointer_cast(cstr);
		hipMemcpy( raw, s.c_str(), cstr_len+1, hipMemcpyHostToDevice );
	}

	// Default constructor.
	__host__ __device__ device_string()
	{
		cstr_len = -1;
		raw = NULL;
	}

	// Conversion operator to copy device_string type to std::string
	// This is where the problem is

	__host__ operator std::string(void)
	{
		std::string ret;
		//device_ptr<char*>::iterator it = cstr.begin();
		thrust::copy(cstr, cstr+cstr_len, back_inserter(ret));
		return ret;
	}
};

char* device_string::pool_raw;
thrust::device_ptr<char> device_string::pool_cstr;
thrust::device_ptr<char> device_string::pool_top;

// User-defined comparison operator
bool __device__ operator< (device_string lhs, device_string rhs)
{
	char *l = lhs.raw;
	char *r = rhs.raw;

	for( ; *l && *r && *l==*r; )
	{
	++l;
	++r;
	}
	return *l < *r;
}
////////////////////DEVICE_STRING ENDS

int main(int argc, char *argv[])
{
	char *word = new(char);
	
	if (argc != 2)
	{
		cout << "Usage: bwt_thrust STRING_INPUT" << endl;
		exit(1);
	}
	
	strcpy(word, argv[1]);
	int N = strlen(word);
	int i;

	char *str, *rot;
	
	hipMalloc((void**)&str, sizeof(char) * (N + 1));
	hipMalloc((void**)&rot, sizeof(char) * ((N + 1) * (N + 1)));
	
	thrust::device_ptr<char> strD(str);
	thrust::device_ptr<char> rotD(rot);
	
	thrust::copy(word, word + N, strD);

	//rotation starts
	for (i = 0; i < N; i++)
	{
		thrust::copy(strD + i, strD + N, rotD + (i * N));
		thrust::copy(strD, strD + i, rotD + (i * N) + (N - i));
	}
	//rotation ends

/*
	//sort starts
	thrust::device_vector<device_string> d_vec;
	d_vec.reserve(sizeof(char) * strlen(word));

	for(vector<std::string>::iterator iter = rotD; iter != rotD + N; ++iter)
	{
		device_string d_str(*iter);
		d_vec.push_back(d_str);
	}

	thrust::sort(d_vec.begin(), d_vec.end() );
	//sort ends
*/

	for (i = 0; i < N; i++)
	{
		hipMemcpy(word, rot + (i * N), N, hipMemcpyDeviceToHost);
		cout << word <<endl;
	}
	
	hipFree(str);
	hipFree(rot);
	
	return 0;
}

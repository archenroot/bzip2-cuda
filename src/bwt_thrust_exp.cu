#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/fill.h>

#include <cstdio>
#include <iostream>
#include <cstring>
#include <iterator>

#define POOL_SZ (10*1024*1024)

using namespace std;

////////////////////DEVICE_STRING STARTS
class device_string
{
public:
	int cstr_len;
	char* raw;
	thrust::device_ptr<char> cstr;

	static char* pool_raw;
	static thrust::device_ptr<char> pool_cstr;
	static thrust::device_ptr<char> pool_top;

	// Sets the variables up the first time its used.
	__host__ static void init()
	{
    		static bool v = true;
		if( v )
		{
			v = false;

			pool_cstr = thrust::device_malloc(POOL_SZ);
			pool_raw  = (char*)raw_pointer_cast( pool_cstr );
			pool_top = pool_cstr;
		}
        }
	// Destructor for device variables used.
	__host__ static void fin()
	{
		init();
		thrust::device_free(pool_cstr);
	}

	// Parametrized constructor to copy one device_string to another.
	__host__ device_string( const device_string& s )
	{
		cstr_len = s.cstr_len;
		raw = s.raw;
		cstr = s.cstr;
	}

	// Parametrized constructor to copy a std::string to device_string type
	__host__ device_string( const std::string& s )
	{
		cstr_len = s.length();
		init();
		cstr = pool_top;
		pool_top += cstr_len+1;
		raw = (char *) raw_pointer_cast(cstr);
		hipMemcpy( raw, s.c_str(), cstr_len+1, hipMemcpyHostToDevice );
	}

	// Default constructor.
	__host__ __device__ device_string()
	{
		cstr_len = -1;
		raw = NULL;
	}

	// Conversion operator to copy device_string type to std::string
	// This is where the problem is

	__host__ operator std::string(void)
	{
		std::string ret;
		//device_ptr<char*>::iterator it = cstr.begin();
		thrust::copy(cstr, cstr+cstr_len, back_inserter(ret));
		return ret;
	}
};

char* device_string::pool_raw;
thrust::device_ptr<char> device_string::pool_cstr;
thrust::device_ptr<char> device_string::pool_top;

// User-defined comparison operator
bool __device__ operator< (device_string lhs, device_string rhs)
{
	char *l = lhs.raw;
	char *r = rhs.raw;

	for( ; *l && *r && *l==*r; )
	{
	++l;
	++r;
	}
	return *l < *r;
}
////////////////////DEVICE_STRING ENDS

void rotate(int N, char *word, vector<string> h_vec)
{
	char *str, *rot;
	hipMalloc((void**)&str, sizeof(char) * (N + 1));
	hipMalloc((void**)&rot, sizeof(char) * ((N + 1) * (N + 1)));
		
	thrust::device_ptr<char> strD(str);
	thrust::device_ptr<char> rotD(rot);
	thrust::copy(word, word + N, strD);
	
	for (int i = 0; i < N; i++)	//Rotations happen in this loop
	{
		thrust::copy(strD + i, strD + N, rotD + (i * N));
		thrust::copy(strD, strD + i, rotD + (i * N) + (N - i));
	}
	
	for (int i = 0; i < N; i++)	//We extract data back from the GPU
	{
		hipMemcpy(word, rot + (i * N), N, hipMemcpyDeviceToHost);
		h_vec.push_back(word);
	}
	
	hipFree(str);
	hipFree(rot);
}

void sort(vector<string> h_vec, char *result)
{
	thrust::device_vector<device_string> d_vec;
	d_vec.reserve(h_vec.size());

	for(vector<std::string>::iterator iter = h_vec.begin(); iter!=h_vec.end(); ++iter)
	{
		device_string d_str(*iter);
		d_vec.push_back(d_str);
	}

	thrust::sort(d_vec.begin(), d_vec.end() );
	
	for(int i = 0; i < d_vec.size(); i++)
	{
		device_string d_str(d_vec[i]);
		h_vec[i] = d_str;
		//cout << h_vec[i] <<endl;
		result[i] = h_vec[i][h_vec[i].length()-1];
	}
}

void bwt( char *word)
{
	int N = strlen(word);
	vector<string> h_vec;
	char *result = new char(N);

	rotate(N, word, h_vec);
	
	sort(h_vec, result);	
	
	cout << result << endl;
}

int main(int argc, char *argv[])
{	
	if (argc != 2)
	{
		cout << "Usage: bwt_thrust STRING_INPUT" << endl;
		exit(1);
	}

	char *word = new(char);	
	strcpy(word, argv[1]);
	bwt(word);
	
	return 0;
}
